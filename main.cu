#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "clib/ndarray.h"
#include "clib/linalg.h"
#include "cudalib/cudalinalg.cuh"

float HOST_TOT_TIME = 0;
float DEVICE_TOT_TIME = 0;

/**
 * Host PCA version
*/
ndarray* PCA(ndarray* M, int k)
{
    ndarray* MT = matTranspose(M);

    ndarray* cov = matProduct(MT, M);
    free_(MT);

    ndarray* E = eigenvectors(cov, k, 1e-10, 1000);

    ndarray* mpca = matProduct(M, E);
    free_(E);

    return mpca;
}

/**
 * Cuda PCA version
*/
ndarray* cudaPCA(ndarray* M, int k)
{
    ndarray* MT = cudaMTranspose(M, 1, false);

    ndarray* cov = cudaMMProduct(MT, M, false);
    cudaFreeHost_(MT);

    ndarray* E = cudaEigenvectors(cov, k, 1e-10, 1000);

    ndarray* mpca = cudaMMProduct(M, E, false);
    cudaFreeHost_(E);

    return mpca;
}

void from_file_example(){

    ndarray* M = new_ndarray(569, 30);

    csv2ndarry(M, "./DATA/breast_cancer.csv", ",");

    printShape(M);

    ndarray* Mpca = PCA(M, 15);
    printShape(Mpca);

    ndarray2csv("out.csv", Mpca, ",");
}

/**
 * Random initializer for tests
*/
void random_init(ndarray* A)
{
    srand(time(NULL));

    for (int i = 0; i < A->shape[0] * A->shape[1]; i++) 
    {
        A->data[i] = (float)rand() / RAND_MAX;
    }
}

/**
 * PCA test
*/
float PCA_TEST(ndarray* A, ndarray* B)
{
    if(A->shape[0] != B->shape[0] || A->shape[1] != B->shape[1])
    {
        printf("ERROR:: File: %s, Line: %d, Function name: PCA_TEST, ", __FILE__, __LINE__);
        printf("reason: %d != %d || %d != %d; A and B must have the same size.\n", A->shape[0], B->shape[0], A->shape[1], B->shape[1]);
        exit(EXIT_FAILURE); 
    }

    int n = A->shape[0] * A->shape[1];
    double err = 0.f;

    for(int i = 0; i < n; i++)
    {
        err += (A->data[i] - B->data[i]) * (A->data[i] - B->data[i]);
    }

    return err == 0 ? 0 : sqrt(err / n);
}

int main(){

    int m = 1 << 20;

    int n = (int) sqrt(m);

    ndarray* M = cuda_ndarrayHost(n, n);
   
    random_init(M);

    ndarray* pca = PCA(M, 20);
    ndarray* cuda_pca = cudaPCA(M, 20);

    float err = PCA_TEST(pca, cuda_pca);
    
    cudaFreeHost_(M);
    free_(pca);
    cudaFreeHost_(cuda_pca);

    float speedup = HOST_TOT_TIME / (DEVICE_TOT_TIME / 1000);

    printf("TEST: PCA vs cudaPCA,\tNumber of elements: %d,\tsize (MB): %.2f,\ttime (s): %.4f vs %.4f, \tGPU speedup: %.4f,\tExpected error: %f\n", m,  m * sizeof(float) / pow(2, 20), HOST_TOT_TIME, DEVICE_TOT_TIME / 1000, speedup, err);
    

    return 0;
}